
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU() {
    printf("Hello, GPU!\n");
}

extern "C" void hello() {
    helloFromGPU<<<1,1>>>();
    hipDeviceSynchronize();
}
